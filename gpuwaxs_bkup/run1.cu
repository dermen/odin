#include "hip/hip_runtime.h"
#include <main.cu>
#include <stdio.h>
#include <vector>
#include <assert.h>
#include <iostream>
#include <fstream>
#include <sstream>

using namespace std;

void uniform_01(vector<float> &f) {
    for(int i=0; i<f.size(); i++) {
        f[i]=(float) rand()/(float) RAND_MAX;
    }
}

float uniform_rand01() {
    


    return 500*(float) rand()/(float) RAND_MAX;
}

void load_in_rands(vector<float> &r1, 
                   vector<float> &r2, 
                   vector<float> &r3, 
                   char *filename) {

    ifstream in(filename);   

    string line;
    int count = 0;
    while(getline(in, line)) {
        stringstream ss(line);
        ss >> r1[count];
        ss >> r2[count];
        ss >> r3[count];
        count++;
    }

    assert(count == r1.size());

    in.close();

}


void load_in_r(vector<float> &rx,
               vector<float> &ry,
               vector<float> &rz,
               vector<int> &id,
               char *filename) {

    ifstream in(filename);   

    string line;
    int count = 0;
    while(getline(in, line)) {
        stringstream ss(line);
        ss >> rx[count];
        ss >> ry[count];
        ss >> rz[count];
        ss >> id[count];
        count++;
    }

    assert(count == rx.size());

    in.close();
}

void load_in_q(vector<float> &qx, 
               vector<float> &qy,
               vector<float> &qz,
               char *filename) {
    ifstream in(filename);   

    string line;
    int count = 0;
    while(getline(in, line)) {
        stringstream ss(line);
        ss >> qx[count];
        ss >> qy[count];
        ss >> qz[count];
        //ss >> oq[count];
        count++;
    }


    assert(count == qx.size());

    in.close();
}


void deviceMalloc( void ** ptr, int bytes) {
    hipError_t err = hipMalloc(ptr, (size_t) bytes);
    assert(err == 0);
} 


inline ostream& operator<< (ostream &out, const vector<int> &s) {
    for(int i=0; i<s.size(); i++) {
        out << s[i] << " ";
    }
    out << endl;
    return out;
}


inline ostream& operator<< (ostream &out, const vector<float> &s) {
    for(int i=0; i<s.size(); i++) {
        out << s[i] << " ";
    }
    out << endl;   
    return out;
}

int main() {

    // allocate qx, qy, qz, outQ, |nQ|
    // allocate rx, ry, rz, |numAtoms|
    // allocate atomicIdentities

    const int tpb = 512;
    const int bpg = 4;

    int nRotations = tpb*bpg;
    int nAtoms = 512; 
    int nQ = 512;

    unsigned int nQ_size = nQ*sizeof(int);
    unsigned int nAtoms_size = nAtoms*sizeof(int);
    unsigned int nRotations_size = nRotations*sizeof(int);

    vector<float> h_qx(nQ);
    vector<float> h_qy(nQ);
    vector<float> h_qz(nQ);
    vector<float> h_outQ(nQ);
    vector<float> h_rx(nAtoms);
    vector<float> h_ry(nAtoms);
    vector<float> h_rz(nAtoms);
    vector<int>   h_id(nAtoms);

    vector<float> h_rand1(nRotations);
    vector<float> h_rand2(nRotations);
    vector<float> h_rand3(nRotations);

    load_in_r(h_rx, h_ry, h_rz, h_id, "512_atom_benchmark.xyz");
    load_in_q(h_qx, h_qy, h_qz, "512_q.xyz");
    load_in_rands(h_rand1, h_rand2, h_rand3, "2048_x_3_random_floats.txt");

    for(int i=0; i < h_outQ.size(); i++) {
        h_outQ[i]=0;
    } 

    float *d_qx; deviceMalloc( (void **) &d_qx, nQ_size);
    float *d_qy; deviceMalloc( (void **) &d_qy, nQ_size);
    float *d_qz; deviceMalloc( (void **) &d_qz, nQ_size);
    float *d_outQ; deviceMalloc( (void **) &d_outQ, nQ_size);
    float *d_rx; deviceMalloc( (void **) &d_rx, nAtoms_size);
    float *d_ry; deviceMalloc( (void **) &d_ry, nAtoms_size);
    float *d_rz; deviceMalloc( (void **) &d_rz, nAtoms_size);
    int   *d_id; deviceMalloc( (void **) &d_id, nAtoms_size);
    float *d_rand1; deviceMalloc( (void **) &d_rand1, nRotations_size);
    float *d_rand2; deviceMalloc( (void **) &d_rand2, nRotations_size);
    float *d_rand3; deviceMalloc( (void **) &d_rand3, nRotations_size);

    hipMemcpy(d_qx, &h_qx[0], nQ_size, hipMemcpyHostToDevice);
    hipMemcpy(d_qy, &h_qy[0], nQ_size, hipMemcpyHostToDevice);
    hipMemcpy(d_qz, &h_qz[0], nQ_size, hipMemcpyHostToDevice);
    hipMemcpy(d_outQ, &h_outQ[0], nQ_size, hipMemcpyHostToDevice);
    hipMemcpy(d_rx, &h_rx[0], nAtoms_size, hipMemcpyHostToDevice);
    hipMemcpy(d_ry, &h_ry[0], nAtoms_size, hipMemcpyHostToDevice);
    hipMemcpy(d_rz, &h_rz[0], nAtoms_size, hipMemcpyHostToDevice);
    hipMemcpy(d_id, &h_id[0], nAtoms_size, hipMemcpyHostToDevice);
    hipMemcpy(d_rand1, &h_rand1[0], nRotations_size, hipMemcpyHostToDevice);
    hipMemcpy(d_rand2, &h_rand2[0], nRotations_size, hipMemcpyHostToDevice);
    hipMemcpy(d_rand3, &h_rand3[0], nRotations_size, hipMemcpyHostToDevice);

    hipError_t error;

    kernel<tpb> <<<bpg, tpb>>> (d_qx, d_qy, d_qz, d_outQ, nQ, d_rx, d_ry, d_rz, d_id, nAtoms, d_rand1, d_rand2, d_rand3);

    hipDeviceSynchronize();
    error = hipGetLastError(); printf("Last error: %d \n", error);

    hipMemcpy(&h_outQ[0], d_outQ, nQ_size, hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    error = hipGetLastError(); printf("Last error: %d \n", error);

    for(int i=0; i<min((int)h_outQ.size(),512); i++) {
        printf("%.0f ", h_outQ[i]);
    }

}
