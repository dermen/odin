/*
This file implements a class that provides an interface for the GPU
scattering code (interface in gpuscatter.hh). It that takes data in on the 
cpu side, copies it to the gpu, and exposes functions that let you perform 
actions with the GPU.

This class will get translated into python via swig
*/


#include <stdio.h>
#include <assert.h>
#include <fstream>
#include <sstream>

#include <gpuscatter.cu>
#include <gpu_mgr.hh>

using namespace std;


void deviceMalloc( void ** ptr, int bytes) {
    hipError_t err = hipMalloc(ptr, (size_t) bytes);
    assert(err == 0);
}


GPUScatter::GPUScatter (int bpg_,      // <-- defines the number of rotations

                        int nQ_,
                        float* h_qx_,    // size: nQ
                        float* h_qy_,    // size: nQ
                        float* h_qz_,    // size: nQ

                        int nAtoms_,
                        float* h_rx_,    // size: nAtoms
                        float* h_ry_,    // size: nAtoms
                        float* h_rz_,    // size: nAtoms
                        float* h_id_,    // size: nAtoms
                        
                        float* h_rand1_, // size: nRotations
                        float* h_rand2_, // size: nRotations
                        float* h_rand3_, // size: nRotations
                        
                        float* h_outQ_   // size: nQ (OUTPUT)
                        ) {
                            
    /* All arguments consist of 
     *   (1) a float pointer to the beginning of the array to be passed
     *   (2) ints representing the size of each array
     */
    
    // unpack arguments
    bpg = bpg_;

    nQ = nQ_;
    h_qx = h_qx_;
    h_qy = h_qy_;
    h_qz = h_qz_;

    nAtoms = nAtoms_;
    h_rx = h_rx_;
    h_ry = h_ry_;
    h_rz = h_rz_;
    h_id = h_id_;

    h_rand1 = h_rand1_;
    h_rand2 = h_rand2_;
    h_rand3 = h_rand3_;

    h_outQ = h_outQ_;
    
    // set some size parameters
    int tpb = 512;
    int nRotations = tpb*bpg;
    
    // compute the memory necessary to hold input/output
    const unsigned int nQ_size = nQ*sizeof(float);
    const unsigned int nAtoms_size = nAtoms*sizeof(float);
    const unsigned int nAtoms_idsize = nAtoms*sizeof(int);
    const unsigned int nRotations_size = nRotations*sizeof(float);

    // allocate memory on the board
    float *d_qx;    deviceMalloc( (void **) &d_qx, nQ_size);
    float *d_qy;    deviceMalloc( (void **) &d_qy, nQ_size);
    float *d_qz;    deviceMalloc( (void **) &d_qz, nQ_size);
    float *d_outQ;  deviceMalloc( (void **) &d_outQ, nQ_size);
    float *d_rx;    deviceMalloc( (void **) &d_rx, nAtoms_size);
    float *d_ry;    deviceMalloc( (void **) &d_ry, nAtoms_size);
    float *d_rz;    deviceMalloc( (void **) &d_rz, nAtoms_size);
    int   *d_id;    deviceMalloc( (void **) &d_id, nAtoms_idsize);
    float *d_rand1; deviceMalloc( (void **) &d_rand1, nRotations_size);
    float *d_rand2; deviceMalloc( (void **) &d_rand2, nRotations_size);
    float *d_rand3; deviceMalloc( (void **) &d_rand3, nRotations_size);

    // copy input/output arrays to board memory
    hipMemcpy(d_qx, &h_qx[0], nQ_size, hipMemcpyHostToDevice);
    hipMemcpy(d_qy, &h_qy[0], nQ_size, hipMemcpyHostToDevice);
    hipMemcpy(d_qz, &h_qz[0], nQ_size, hipMemcpyHostToDevice);
    hipMemcpy(d_outQ, &h_outQ[0], nQ_size, hipMemcpyHostToDevice);
    hipMemcpy(d_rx, &h_rx[0], nAtoms_size, hipMemcpyHostToDevice);
    hipMemcpy(d_ry, &h_ry[0], nAtoms_size, hipMemcpyHostToDevice);
    hipMemcpy(d_rz, &h_rz[0], nAtoms_size, hipMemcpyHostToDevice);
    hipMemcpy(d_id, &h_id[0], nAtoms_idsize, hipMemcpyHostToDevice);
    hipMemcpy(d_rand1, &h_rand1[0], nRotations_size, hipMemcpyHostToDevice);
    hipMemcpy(d_rand2, &h_rand2[0], nRotations_size, hipMemcpyHostToDevice);
    hipMemcpy(d_rand3, &h_rand3[0], nRotations_size, hipMemcpyHostToDevice);

    // check for errors
    hipError_t err = hipGetLastError();
    assert(err == 0);  
}

void GPUScatter::run() {
    // execute the kernel
    kernel<tpb> <<<bpg, tpb>>> (d_qx, d_qy, d_qz, d_outQ, nQ, d_rx, d_ry, d_rz, d_id, nAtoms, d_rand1, d_rand2, d_rand3);
    hipDeviceSynchronize();
    hipError_t err = hipGetLastError();
    assert(err == 0);
}

void GPUScatter::retreive() {
    // retrieve the output off the board and back into CPU memory
    // copys the array to the output array passed as input
    hipMemcpy(&h_outQ[0], d_outQ, nQ_size, hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    hipError_t err = hipGetLastError();
    assert(err == 0);
}

GPUScatter::~GPUScatter() {
    // destroy the class
    hipFree(d_qx);
    hipFree(d_qy);
    hipFree(d_qz);
    hipFree(d_rx);
    hipFree(d_ry);
    hipFree(d_rz);
    hipFree(d_id);
    hipFree(d_rand1);
    hipFree(d_rand2);
    hipFree(d_rand3);
    hipFree(d_outQ);
}
