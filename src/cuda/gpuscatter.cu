#include "hip/hip_runtime.h"
/*! YTZ 20121106 */
#include <stdio.h>
#include <stdint.h>

// warning: this code is not safe due to reduction if total # of threads != multiple of
// blockSize ... too lazy to add in ifs for now 
// todo: add in ifs and while loops for > 67million
void __device__ generate_random_quaternion(real r1, real r2, real r3,
                real &q1, real &q2, real &q3, real &q4) {
    
    real s, sig1, sig2, theta1, theta2, w, x, y, z;
    
    s = r1;
    sig1 = sqrt(s);
    sig2 = sqrt(1.0 - s);
    
    theta1 = 2.0 * M_PI * r2;
    theta2 = 2.0 * M_PI * r3;
    
    w = cos(theta2) * sig2;
    x = sin(theta1) * sig1;
    y = cos(theta1) * sig1;
    z = sin(theta2) * sig2;
    
    q1 = w;
    q2 = x;
    q3 = y;
    q4 = z;
}

__device__ double atomicAdd(double* address, double val) {
    double old = *address, assumed;
    do{
        assumed = old;
        old =__longlong_as_double(atomicCAS((unsigned long long int*)address,
            __double_as_longlong(assumed),
            __double_as_longlong(val + assumed)));
    }
    while(assumed != old);
    return old;
}


void __device__ rotate(real x, real y, real z,
                       real b0, real b1, real b2, real b3,
                       real &ox, real &oy, real &oz) {

    // x,y,z      -- real vector
    // b          -- quaternion for rotation
    // ox, oy, oz -- rotated real vector
    
    real a0 = 0;
    real a1 = x;
    real a2 = y;
    real a3 = z;

    real c0 = b0*a0 - b1*a1 - b2*a2 - b3*a3;
    real c1 = b0*a1 + b1*a0 + b2*a3 - b3*a2;
    real c2 = b0*a2 - b1*a3 + b2*a0 + b3*a1;
    real c3 = b0*a3 + b1*a2 - b2*a1 + b3*a0;   

    real bb0 = b0;
    real bb1 = -b1;
    real bb2 = -b2;
    real bb3 = -b3;

  //real cc0 = c0*bb0 - c1*bb1 - c2*bb2 - c3*bb3;
    real cc1 = c0*bb1 + c1*bb0 + c2*bb3 - c3*bb2;
    real cc2 = c0*bb2 - c1*bb3 + c2*bb0 + c3*bb1;
    real cc3 = c0*bb3 + c1*bb2 - c2*bb1 + c3*bb0;   

    ox = cc1;
    oy = cc2;
    oz = cc3;

}


template<unsigned int blockSize>
void __global__ kernel(real const * const __restrict__ q_x, 
                       real const * const __restrict__ q_y, 
                       real const * const __restrict__ q_z, 
                       real *outQ, // <-- not const 
                       int   const nQ,
		               real const * const __restrict__ r_x, 
                       real const * const __restrict__ r_y, 
                       real const * const __restrict__ r_z,
		               int   const * const __restrict__ atomicIdentities, 
                       int   const numAtoms, 
                       real const * const __restrict__ randN1, 
                       real const * const __restrict__ randN2, 
                       real const * const __restrict__ randN3) {
    // shared array for block-wise reduction
    __shared__ real sdata[blockSize];
    
    int tid = threadIdx.x;
	int gid = blockIdx.x*blockDim.x + threadIdx.x;

    // determine the rotated locations
    real rand1 = randN1[gid]; 
    real rand2 = randN2[gid]; 
    real rand3 = randN3[gid]; 

    // rotation quaternions
    real q0, q1, q2, q3;
    generate_random_quaternion(rand1, rand2, rand3, q0, q1, q2,q3);

    // for each q vector
    for(int iq = 0; iq < nQ; iq++) {
        real qx = q_x[iq];
        real qy = q_y[iq];
        real qz = q_z[iq];
        real mq = qx*qx+qy*qy+qz*qz;
        real qo = mq / (4*4*M_PI*M_PI);
        //accumulant
        real2 Qsum;
        Qsum.x = 0;
        Qsum.y = 0;
        // for each atom in molecule

        // precompute fis
        real fi1, fi79;
        fi1=fi79=0;

        // if H
        fi1  = 0.493002*exp(-10.5109*qo);
        fi1 += 0.322912*exp(-26.1257*qo);
        fi1 += 0.140191*exp(-3.14236*qo);
        fi1 += 0.040810*exp(-57.7997*qo);
        fi1 += 0.003038;
        // if Au
        fi79  = 16.8819*exp(-0.4611*qo);
        fi79 += 18.5913*exp(-8.6216*qo);
        fi79 += 25.5582*exp(-1.4826*qo);
        fi79 += 5.86*exp(-36.3956*qo);
        fi79 += 12.0658; 
        /*
        // if C
        fi8  = 3.04850*exp(-13.2771*qo);
        fi8 += 2.28680*exp(-5.70110*qo);
        fi8 += 1.54630*exp(-0.323900*qo);
        fi8 += 0.867000*exp(-32.9089*qo);
        fi8 += 0.2508;
        // if N 
        fi7  = 12.2126*exp(-0.005700*qo);
        fi7 += 3.13220*exp(-9.89330*qo);
        fi7 += 2.01250*exp(-28.9975*qo);
        fi7 += 1.16630*exp(-0.582600*qo);
        fi7 += -11.529;
         // if Fe
        fi26  = 11.7695*exp(-4.7611*qo);
        fi26 += 7.35730*exp(-0.307200*qo);
        fi26 += 3.52220*exp(-15.3535*qo);
        fi26 += 2.30450*exp(-76.8805*qo);
        fi26 += 1.03690;
        // else default to N
        fid  = 12.2126*exp(-0.005700*qo);
        fid += 3.13220*exp(-9.89330*qo);
        fid += 2.01250*exp(-28.9975*qo);
        fid += 1.16630*exp(-0.582600*qo);
        fid += -11.529;
        */

        for(int a = 0; a < numAtoms; a++) {
            // calculate fi
            real fi = 0;
            int atomicNumber = atomicIdentities[a];
            if(atomicNumber == 1) {
                fi = fi1;
            } else if(atomicNumber == 79) {
                fi = fi79;
            // else default to N
            } 
            // get the current positions
            real rx = r_x[a];
            real ry = r_y[a];
            real rz = r_z[a];
            real ax, ay, az;

            rotate(rx, ry, rz, q0, q1, q2, q3, ax, ay, az);
            real qr = ax*qx + ay*qy + az*qz;

            Qsum.x += fi*__sinf(qr);
            Qsum.y += fi*__cosf(qr);
            
        } // finished one molecule.
        real fQ = Qsum.x*Qsum.x + Qsum.y*Qsum.y;  
        sdata[tid] = fQ;
        __syncthreads();
        // Todo: quite slow but correct, speed up reduction later if becomes bottleneck!
        for(unsigned int s=1; s < blockDim.x; s *= 2) {
            if(tid % (2*s) == 0) {
                sdata[tid] += sdata[tid+s];
            }
            __syncthreads();
        }
        if(tid == 0) {
            atomicAdd(outQ+iq, sdata[0]); 
        } 
    }
}

__global__ void randTest(real *a) {
    int gid = blockIdx.x*blockDim.x + threadIdx.x;

    int tt = __cosf(gid);
    int yy = __sinf(gid);

    a[gid] = tt;
    a[gid/2] = yy;
}
