#include "hip/hip_runtime.h"
/*! YTZ 20121106 */
#include <stdio.h>
#include <stdint.h>

// warning: this code is not safe due to reduction if total # of threads != multiple of
// blockSize ... too lazy to add in ifs for now 
// todo: add in ifs and while loops for > 67million
void __device__ generate_random_quaternion(float r1, float r2, float r3,
                float &q1, float &q2, float &q3, float &q4) {
    
    float s, sig1, sig2, theta1, theta2, w, x, y, z;
    
    s = r1;
    sig1 = sqrt(s);
    sig2 = sqrt(1.0 - s);
    
    theta1 = 2.0 * M_PI * r2;
    theta2 = 2.0 * M_PI * r3;
    
    w = cos(theta2) * sig2;
    x = sin(theta1) * sig1;
    y = cos(theta1) * sig1;
    z = sin(theta2) * sig2;
    
    q1 = w;
    q2 = x;
    q3 = y;
    q4 = z;
}

__device__ double atomicAdd(double* address, double val) {
    double old = *address, assumed;
    do{
        assumed = old;
        old =__longlong_as_double(atomicCAS((unsigned long long int*)address,
            __double_as_longlong(assumed),
            __double_as_longlong(val + assumed)));
    }
    while(assumed != old);
    return old;
}


void __device__ rotate(float x, float y, float z,
                       float b0, float b1, float b2, float b3,
                       float &ox, float &oy, float &oz) {

    // x,y,z      -- float vector
    // b          -- quaternion for rotation
    // ox, oy, oz -- rotated float vector
    
    float a0 = 0;
    float a1 = x;
    float a2 = y;
    float a3 = z;

    float c0 = b0*a0 - b1*a1 - b2*a2 - b3*a3;
    float c1 = b0*a1 + b1*a0 + b2*a3 - b3*a2;
    float c2 = b0*a2 - b1*a3 + b2*a0 + b3*a1;
    float c3 = b0*a3 + b1*a2 - b2*a1 + b3*a0;   

    float bb0 = b0;
    float bb1 = -b1;
    float bb2 = -b2;
    float bb3 = -b3;

  //float cc0 = c0*bb0 - c1*bb1 - c2*bb2 - c3*bb3;
    float cc1 = c0*bb1 + c1*bb0 + c2*bb3 - c3*bb2;
    float cc2 = c0*bb2 - c1*bb3 + c2*bb0 + c3*bb1;
    float cc3 = c0*bb3 + c1*bb2 - c2*bb1 + c3*bb0;   

    ox = cc1;
    oy = cc2;
    oz = cc3;

}


template<unsigned int blockSize>
void __global__ kernel(float const * const __restrict__ q_x, 
                       float const * const __restrict__ q_y, 
                       float const * const __restrict__ q_z, 
                       float *outQ, // <-- not const 
                       int   const nQ,
		               float const * const __restrict__ r_x, 
                       float const * const __restrict__ r_y, 
                       float const * const __restrict__ r_z,
		               float const * const __restrict__ frmfcts, 
                       int   const numAtoms, 
                       float const * const __restrict__ randN1, 
                       float const * const __restrict__ randN2, 
                       float const * const __restrict__ randN3) {

    // shared array for block-wise reduction
    __shared__ float sdata[blockSize];
    
    int tid = threadIdx.x;
    int gid = blockIdx.x*blockDim.x + threadIdx.x;

    // determine the rotated locations
    float rand1 = randN1[gid]; 
    float rand2 = randN2[gid]; 
    float rand3 = randN3[gid]; 

    // rotation quaternions
    float q0, q1, q2, q3;
    generate_random_quaternion(rand1, rand2, rand3, q0, q1, q2,q3);

    // for each q vector
    for(int iq = 0; iq < nQ; iq++) {
        float qx = q_x[iq];
        float qy = q_y[iq];
        float qz = q_z[iq];
        float mq = qx*qx+qy*qy+qz*qz;
        float qo = mq / (4*4*M_PI*M_PI);

        //accumulant
        float2 Qsum;
        Qsum.x = 0;
        Qsum.y = 0;

        // for each atom in molecule
        for(int a = 0; a < numAtoms; a++) {

            // retrieve the atomic form factor
            // TJL modified this to pass in the atomic form factors from python
            fi = frmfcts[a];

            // get the current positions
            float rx = r_x[a];
            float ry = r_y[a];
            float rz = r_z[a];
            float ax, ay, az;

            rotate(rx, ry, rz, q0, q1, q2, q3, ax, ay, az);
            float qr = ax*qx + ay*qy + az*qz;

            Qsum.x += fi*__sinf(qr);
            Qsum.y += fi*__cosf(qr);
            
        } // finished one molecule.
        float fQ = Qsum.x*Qsum.x + Qsum.y*Qsum.y;  
        sdata[tid] = fQ;
        __syncthreads();
        // Todo: quite slow but correct, speed up reduction later if becomes bottleneck!
        for(unsigned int s=1; s < blockDim.x; s *= 2) {
            if(tid % (2*s) == 0) {
                sdata[tid] += sdata[tid+s];
            }
            __syncthreads();
        }
        if(tid == 0) {
            atomicAdd(outQ+iq, sdata[0]); 
        } 
    }
}

__global__ void randTest(float *a) {
    int gid = blockIdx.x*blockDim.x + threadIdx.x;

    int tt = __cosf(gid);
    int yy = __sinf(gid);

    a[gid] = tt;
    a[gid/2] = yy;
}
