/*
This file implements a class that provides an interface for the GPU
scattering code (interface in gpuscatter.hh). It that takes data in on the 
cpu side, copies it to the gpu, and exposes functions that let you perform 
actions with the GPU.

This class will get translated into python via swig
*/


#include <stdio.h>
#include <assert.h>
#include <fstream>
#include <sstream>

#include <gpuscatter.cu>
#include <gpuscatter_mgr.hh>

using namespace std;


void deviceMalloc( void ** ptr, int bytes) {
    hipError_t err = hipMalloc(ptr, (size_t) bytes);
    assert(err == 0);
}


GPUScatter::GPUScatter (int bpg_,      // <-- defines the number of rotations
            
                        // scattering q-vectors
                        int    nQx_,
                        int    nQy_,
                        int    nQz_,
                        float* h_qx_,
                        float* h_qy_,
                        float* h_qz_,
                
                        // atomic positions, ids
                        int    nAtomsx_,
                        int    nAtomsy_,
                        int    nAtomsz_,
                        float* h_rx_,
                        float* h_ry_,
                        float* h_rz_,
                        int*   h_id_,

                        // cromer-mann parameters
                        int    nCM_,
                        float* h_cm_,

                        // random numbers for rotations
                        int    nRot1_,
                        int    nRot2_,
                        int    nRot3_,
                        float* h_rand1_,
                        float* h_rand2_,
                        float* h_rand3_,

                        // output
                        int    nQout_,
                        float* h_outQ_
                        ) {
                            
    /* All arguments consist of 
     *   (1) a float pointer to the beginning of the array to be passed
     *   (2) ints representing the size of each array
     */
     
    // many of the arrays above are 1D arrays that should be the same len
    // due to the SWIG wrapping, however, we had to pass each individually
    // so now check that they are, in fact, the correct dimension
    assert( nQx_ == nQy_ )
    assert( nQx_ == nQz_ )
    assert( nQx_ == nQout_ )
    
    assert( nAtomsx_ == nAtomsy_ )
    assert( nAtomsx_ == nAtomsz_ )
    
    assert( nRot1_ == nRot2_ )
    assert( nRot1_ == nRot3_ )
    
    assert( bpg_ / 512 == nRot1_ )
    assert( nRot1_ == nRot2_ )
    assert( nRot1_ == nRot3_ )
    
    
    // unpack arguments
    bpg = bpg_;

    nQ = nQx_;
    h_qx = h_qx_;
    h_qy = h_qy_;
    h_qz = h_qz_;

    nAtoms = nAtomsx_;
    numAtomTypes = nCM_ / 9;
    h_rx = h_rx_;
    h_ry = h_ry_;
    h_rz = h_rz_;
    h_id = h_id_;

    h_cm = h_cm_;

    h_rand1 = h_rand1_;
    h_rand2 = h_rand2_;
    h_rand3 = h_rand3_;

    h_outQ = h_outQ_;
    
    // set some size parameters
    static const int tpb = 512;
    int nRotations = tpb*bpg;
    
    // compute the memory necessary to hold input/output
    const unsigned int nQ_size = nQ*sizeof(float);
    const unsigned int nAtoms_size = nAtoms*sizeof(float);
    const unsigned int nAtoms_idsize = nAtoms*sizeof(int);
    const unsigned int nRotations_size = nRotations*sizeof(float);
    const unsigned int cm_size = 9*numAtomTypes*sizeof(float);

    // allocate memory on the board
    float *d_qx;        deviceMalloc( (void **) &d_qx, nQ_size);
    float *d_qy;        deviceMalloc( (void **) &d_qy, nQ_size);
    float *d_qz;        deviceMalloc( (void **) &d_qz, nQ_size);
    float *d_outQ;      deviceMalloc( (void **) &d_outQ, nQ_size);
    float *d_rx;        deviceMalloc( (void **) &d_rx, nAtoms_size);
    float *d_ry;        deviceMalloc( (void **) &d_ry, nAtoms_size);
    float *d_rz;        deviceMalloc( (void **) &d_rz, nAtoms_size);
    int   *d_id;        deviceMalloc( (void **) &d_id, nAtoms_idsize);
    float *d_cm;        deviceMalloc( (void **) &d_cm, cm_size);
    float *d_rand1;     deviceMalloc( (void **) &d_rand1, nRotations_size);
    float *d_rand2;     deviceMalloc( (void **) &d_rand2, nRotations_size);
    float *d_rand3;     deviceMalloc( (void **) &d_rand3, nRotations_size);

    // copy input/output arrays to board memory
    hipMemcpy(d_qx, &h_qx[0], nQ_size, hipMemcpyHostToDevice);
    hipMemcpy(d_qy, &h_qy[0], nQ_size, hipMemcpyHostToDevice);
    hipMemcpy(d_qz, &h_qz[0], nQ_size, hipMemcpyHostToDevice);
    hipMemcpy(d_outQ, &h_outQ[0], nQ_size, hipMemcpyHostToDevice);
    hipMemcpy(d_rx, &h_rx[0], nAtoms_size, hipMemcpyHostToDevice);
    hipMemcpy(d_ry, &h_ry[0], nAtoms_size, hipMemcpyHostToDevice);
    hipMemcpy(d_rz, &h_rz[0], nAtoms_size, hipMemcpyHostToDevice);
    hipMemcpy(d_id, &h_id[0], nAtoms_idsize, hipMemcpyHostToDevice);
    hipMemcpy(d_cm, &h_cm[0], cm_size, hipMemcpyHostToDevice);
    hipMemcpy(d_rand1, &h_rand1[0], nRotations_size, hipMemcpyHostToDevice);
    hipMemcpy(d_rand2, &h_rand2[0], nRotations_size, hipMemcpyHostToDevice);
    hipMemcpy(d_rand3, &h_rand3[0], nRotations_size, hipMemcpyHostToDevice);

    // check for errors
    hipError_t err = hipGetLastError();
    assert(err == 0);  
}

void GPUScatter::run() {
    // execute the kernel
    kernel<tpb> <<<bpg, tpb>>> (d_qx, d_qy, d_qz, d_outQ, nQ, d_rx, d_ry, d_rz, d_id, nAtoms, numAtomTypes, d_cm, d_rand1, d_rand2, d_rand3);
    hipDeviceSynchronize();
    hipError_t err = hipGetLastError();
    assert(err == 0);
}

void GPUScatter::retreive() {
    // retrieve the output off the board and back into CPU memory
    // copys the array to the output array passed as input
    hipMemcpy(&h_outQ[0], d_outQ, nQ_size, hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    hipError_t err = hipGetLastError();
    assert(err == 0);
}

GPUScatter::~GPUScatter() {
    // destroy the class
    hipFree(d_qx);
    hipFree(d_qy);
    hipFree(d_qz);
    hipFree(d_rx);
    hipFree(d_ry);
    hipFree(d_rz);
    hipFree(d_id);
    hipFree(d_cm);
    hipFree(d_rand1);
    hipFree(d_rand2);
    hipFree(d_rand3);
    hipFree(d_outQ);
}
